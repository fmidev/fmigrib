#include "hip/hip_runtime.h"
#include "NFmiGribPacking.h"
#include <cassert>
#include <thrust/device_ptr.h>
#include <thrust/extrema.h>

// custom atomicAdd for unsigned char because cuda libraries do not have it
__device__ unsigned char atomicAdd(unsigned char* address, unsigned char val)
{
	unsigned int* address_as_ui = (unsigned int*)(address - ((size_t)address & 3));
	unsigned int old = *address_as_ui;
	const unsigned int shift = (((size_t)address & 3) * 8);
	unsigned int sum;
	unsigned int assumed;

	do
	{
		assumed = old;
		sum = val + static_cast<unsigned char>((old >> shift) & 0xff);
		old = (old & ~(0x000000ff << shift)) | (sum << shift);
		old = atomicCAS(address_as_ui, assumed, old);
	} while (assumed != old);

	return old;
}

template <typename T>
__global__ void InitializeArrayKernel(T* d_arr, T val, size_t N)
{
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	int stride = blockDim.x * gridDim.x;

	for (; idx < N; idx += stride)
	{
		d_arr[idx] = val;
	}
}

template <typename T>
void InitializeArray(T* d_arr, T val, size_t N, hipStream_t& stream)
{
	const int blockSize = 128;
	const int gridSize = N / blockSize + (N % blockSize == 0 ? 0 : 1);

	InitializeArrayKernel<T><<<gridSize, blockSize, 0, stream>>>(d_arr, val, N);
}

void NFmiGribPacking::UnpackBitmap(const unsigned char* __restrict__ bitmap, int* __restrict__ unpacked, size_t len,
                                   size_t unpackedLen)
{
	size_t i, idx = 0;
	int v = 1;

	short j = 0;

	for (i = 0; i < len; i++)
	{
		for (j = 7; j >= 0; j--)
		{
			if (BitTest(bitmap[i], j))
			{
				unpacked[idx] = v++;
			}
			else
			{
				unpacked[idx] = 0;
			}

			if (++idx >= unpackedLen)
			{
				// packed data might not be aligned nicely along byte boundaries --
				// need to break from loop after final element has been processed
				break;
			}
		}
	}
}

template <typename T>
__host__ T Min(T* d_arr, size_t N, hipStream_t& stream)
{
	T* ret = thrust::min_element(thrust::cuda::par.on(stream), d_arr, d_arr + N);

	return *ret;
}

template <typename T>
__host__ T Max(T* d_arr, size_t N, hipStream_t& stream)
{
	T* ret = thrust::max_element(thrust::cuda::par.on(stream), d_arr, d_arr + N);

	return *ret;
}

long NFmiGribPacking::simple_packing::get_binary_scale_fact(double max, double min, long bpval)
{
	assert(max >= min);
	double range = max - min;
	double zs = 1;
	long scale = 0;
	const long last = 127; /* Depends on edition, should be parameter */

	unsigned long maxint = NFmiGribPacking::ToPower(bpval, 2) - 1;
	double dmaxint = (double)maxint;

	assert(bpval >= 1);

	if (range == 0)
		return 0;

	/* range -= 1e-10; */
	while ((range * zs) <= dmaxint)
	{
		scale--;
		zs *= 2;
	}

	while ((range * zs) > dmaxint)
	{
		scale++;
		zs /= 2;
	}

	while ((unsigned long)(range * zs + 0.5) <= maxint)
	{
		scale--;
		zs *= 2;
	}

	while ((unsigned long)(range * zs + 0.5) > maxint)
	{
		scale++;
		zs /= 2;
	}

	if (scale < -last)
	{
		printf("grib_get_binary_scale_fact: max=%g min=%g\n", max, min);
		scale = -last;
	}
	assert(scale <= last);

	return scale;
}

long NFmiGribPacking::simple_packing::get_decimal_scale_fact(double max, double min, long bpval, long binary_scale)
{
	// Copied from eccodes library
	assert(max >= min);

	double range = max - min;
	const long last = 127; /* Depends on edition, should be parameter */
	double decimal_scale_factor = 0;
	double f;
	double minrange = 0, maxrange = 0;
	double decimal = 1;
	long bits_per_value = bpval;

	double unscaled_min = min;
	double unscaled_max = max;

	f = NFmiGribPacking::ToPower(bits_per_value, 2) - 1;
	minrange = NFmiGribPacking::ToPower(-last, 2.) * f;
	maxrange = NFmiGribPacking::ToPower(last, 2.) * f;

	while (range < minrange)
	{
		decimal_scale_factor += 1;
		decimal *= 10;
		min = unscaled_min * decimal;
		max = unscaled_max * decimal;
		range = (max - min);
	}
	while (range > maxrange)
	{
		decimal_scale_factor -= 1;
		decimal /= 10;
		min = unscaled_min * decimal;
		max = unscaled_max * decimal;
		range = (max - min);
	}

	return decimal_scale_factor;
}

template <typename T>
__device__ void PackUnevenBytes(unsigned char* __restrict__ d_p, const T* __restrict__ d_u,
                                NFmiGribPacking::packing_coefficients coeff, int idx)

{
	const double decimal = NFmiGribPacking::ToPower(-coeff.decimalScaleFactor, 10);
	const double divisor = NFmiGribPacking::ToPower(-coeff.binaryScaleFactor, 2);

	const double x = fma(fma(static_cast<double>(d_u[idx]), decimal, -coeff.referenceValue), divisor, 0.5);
	const unsigned int val = __double2uint_rd(x);

	unsigned int bitp = coeff.bitsPerValue * idx;

	d_p += (bitp / 8);

	unsigned char accum = 0;

	for (int i = coeff.bitsPerValue - 1; i >= 0; i--)
	{
		const int onoff = BitTest(val, i);
		const unsigned char ad = 1 << (7 - (bitp % 8));

		accum += (onoff * ad);
		bitp++;

		if (bitp % 8 == 0)
		{
			atomicAdd(d_p, accum);

			// change of byte (memory location)
			d_p++;
			accum = 0;
		}
	}

	atomicAdd(d_p, accum);
}

template <typename T>
__device__ void PackFullBytes(unsigned char* __restrict__ d_p, const T* __restrict__ d_u,
                              NFmiGribPacking::packing_coefficients coeff, int idx)
{
	const double decimal = NFmiGribPacking::ToPower(-coeff.decimalScaleFactor, 10);
	const double divisor = NFmiGribPacking::ToPower(-coeff.binaryScaleFactor, 2);

	const double x = fma(fma(static_cast<double>(d_u[idx]), decimal, -coeff.referenceValue), divisor, 0.5);
	const unsigned int val = __double2uint_rd(x);

	unsigned char* encoded = &d_p[idx * static_cast<int>(coeff.bitsPerValue / 8)];

	while (coeff.bitsPerValue >= 8)
	{
		coeff.bitsPerValue -= 8;
		*encoded = (val >> coeff.bitsPerValue);
		encoded++;
	}
}

template <typename T>
__global__ void PackSimpleKernel(const T* __restrict__ d_u, unsigned char* __restrict__ d_p, const int* d_b, size_t N,
                                 NFmiGribPacking::packing_coefficients coeff)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < N)
	{
		if (coeff.bitsPerValue % 8)
		{
			PackUnevenBytes(d_p, d_u, coeff, idx);
		}
		else
		{
			PackFullBytes(d_p, d_u, coeff, idx);
		}
	}
}

template <typename T>
bool NFmiGribPacking::simple_packing::Pack(T* arr, unsigned char* packed, const int* d_bitmap, size_t unpackedLen,
                                           NFmiGribPacking::packing_coefficients coeffs, hipStream_t& stream)
{
	// 1. Check pointer type

	bool isHostMemory = IsHostPointer(arr);

	T* d_arr = 0;

	if (!isHostMemory)
	{
		d_arr = arr;
	}

	// 2. Copy unpacked data to device if needed

	if (isHostMemory)
	{
		CUDA_CHECK(hipHostRegister(arr, sizeof(T) * unpackedLen, 0));
		CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&d_arr), unpackedLen * sizeof(T)));
		CUDA_CHECK(hipMemcpyAsync(d_arr, reinterpret_cast<void*>(arr), unpackedLen * sizeof(T), hipMemcpyHostToDevice,
		                           stream));
	}

	unsigned char* d_packed = 0;

	long packedLen = ((coeffs.bitsPerValue * unpackedLen) + 7) / 8;
	CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&d_packed), packedLen * sizeof(unsigned char)));
	InitializeArray<unsigned char>(d_packed, 0u, packedLen, stream);
	CUDA_CHECK_ERROR_MSG("Kernel invocation");

	const int blockSize = 256;
	const int gridSize = unpackedLen / blockSize + (unpackedLen % blockSize == 0 ? 0 : 1);

	PackSimpleKernel<T><<<gridSize, blockSize, 0, stream>>>(d_arr, d_packed, d_bitmap, unpackedLen, coeffs);

	CUDA_CHECK(hipMemcpyAsync(packed, d_packed, packedLen * sizeof(unsigned char), hipMemcpyDeviceToHost, stream));
	CUDA_CHECK(hipStreamSynchronize(stream));
	CUDA_CHECK_ERROR_MSG("Kernel invocation");

	CUDA_CHECK(hipFree(d_packed));

	if (isHostMemory)
	{
		CUDA_CHECK(hipFree(d_arr));
		CUDA_CHECK(hipHostUnregister(arr));
	}

	return true;
}

template bool NFmiGribPacking::simple_packing::Pack(double*, unsigned char*, const int*, size_t,
                                                    NFmiGribPacking::packing_coefficients, hipStream_t&);
template bool NFmiGribPacking::simple_packing::Pack(float*, unsigned char*, const int*, size_t,
                                                    NFmiGribPacking::packing_coefficients, hipStream_t&);
