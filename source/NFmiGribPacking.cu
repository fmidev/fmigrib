#include "hip/hip_runtime.h"
#include "NFmiGribPacking.h"
#include <thrust/device_ptr.h>
#include <thrust/fill.h>
#include <hip/hip_runtime_api.h>

void NFmiGribPacking::Fill(double* arr, size_t len, double fillValue)
{
	thrust::device_ptr<double> ptr = thrust::device_pointer_cast(arr);
	thrust::fill(ptr, ptr + len, fillValue);
}

template <typename T>
bool NFmiGribPacking::IsHostPointer(const T* ptr)
{
	hipPointerAttribute_t attributes;
	hipError_t err = hipPointerGetAttributes(&attributes, ptr);

	bool ret;

	if (err == hipErrorInvalidValue && ptr)
	{
		ret = true;

		// Clear error buffer
		hipGetLastError();
	}
	else if (err == hipSuccess)
	{
#if CUDART_VERSION >= 10010
		if (attributes.type == hipMemoryTypeHost)
#else
		if (attributes.memoryType == hipMemoryTypeHost)
#endif
		{
			ret = true;
		}
		else
		{
			ret = false;
		}
	}
	else
	{
		std::cerr << "simple_packing::Pack Error " << static_cast<int>(err) << " (" << hipGetErrorString(err)
		          << ") while checking pointer attributes" << std::endl;
		exit(1);
	}

	return ret;
}

template bool NFmiGribPacking::IsHostPointer(const double*);
template bool NFmiGribPacking::IsHostPointer(const float*);

template <typename T>
__host__ __device__ void MinMax_(T* d, size_t unpackedLen, T& min, T& max)
{
	using namespace NFmiGribPacking::simple_packing;
	min = MissingValue<T>();
	max = MissingValue<T>();

	for (size_t i = 0; i < unpackedLen; i++)
	{
		T val = d[i];
		if (IsMissing(val))
			continue;

		if (val < min)
			min = val;
		if (val > max)
			max = val;
	}
}

template __host__ __device__ void MinMax_(double*, size_t, double&, double&);
template __host__ __device__ void MinMax_(float*, size_t, float&, float&);

template <typename T>
__global__ void MinMaxKernel(T* d, size_t unpackedLen, T& min, T& max)
{
	MinMax_<T>(d, unpackedLen, min, max);
}

template <typename T>
void NFmiGribPacking::MinMax(T* d, size_t unpackedLen, T& min, T& max, hipStream_t& stream)
{
	if (IsHostPointer<T>(d))
	{
		MinMax_(d, unpackedLen, min, max);
	}
	else
	{
		double* d_min = 0;
		double* d_max = 0;
		CUDA_CHECK(hipMalloc(&d_min, sizeof(double)));
		CUDA_CHECK(hipMalloc(&d_max, sizeof(double)));

		MinMaxKernel<<<1, 1, 0, stream>>>(d, unpackedLen, min, max);

		CUDA_CHECK(hipMemcpyAsync(&min, d_min, sizeof(double), hipMemcpyDeviceToHost, stream));
		CUDA_CHECK(hipMemcpyAsync(&max, d_max, sizeof(double), hipMemcpyDeviceToHost, stream));

		CUDA_CHECK(hipStreamSynchronize(stream));
		CUDA_CHECK(hipFree(d_min));
		CUDA_CHECK(hipFree(d_max));
	}
}

template void NFmiGribPacking::MinMax(double*, size_t, double&, double&, hipStream_t&);
template void NFmiGribPacking::MinMax(float*, size_t, float&, float&, hipStream_t&);
